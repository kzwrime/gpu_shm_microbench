#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>

// Size of array
#define N 256 * 80 * 16

#define WARM_UP_LOOP 200
#define KERNEL_LOOP 100
#define KERNEL_INNER_REPEAT 10000

enum share_read {
  no_conflict,
  boardcast,
  multicast,
  conflict_2_way,
  conflict_4_way,
  conflict_8_way,
  conflict_16_way
};

#define HOST_DEVICE_INLINE __host__ __device__ __forceinline__

HOST_DEVICE_INLINE float2 operator+(const float2 &u, const float2 &v) {
  return make_float2(u.x + v.x, u.y + v.y);
}

HOST_DEVICE_INLINE void operator+=(float2 &u, const float2 &v) {
  u.x += v.x;
  u.y += v.y;
}

HOST_DEVICE_INLINE float4 operator+(const float4 &u, const float4 &v) {
  return make_float4(u.x + v.x, u.y + v.y, u.z + v.z, u.w + v.w);
}

HOST_DEVICE_INLINE void operator+=(float4 &u, const float4 &v) {
  u.x += v.x;
  u.y += v.y;
  u.z += v.z;
  u.w += v.w;
}

#define MTYPE float
#define MAKE_MTYPE(x) (x)
#define GET_MTYPE(x) (x)

// #define MTYPE float2
// #define MAKE_MTYPE(x) make_float2(x, x)
// #define GET_MTYPE(v) (v.x)

// #define MTYPE float4
// #define MAKE_MTYPE(x) make_float4(x, x, x, x)
// #define GET_MTYPE(v) (v.x)

// Kernel
template <int choose> __global__ void add_vectors(float *a) {
  const int ITEMS = 8;
  __shared__ MTYPE shm[ITEMS][256]; // may be no enough for conflict_16_way
  int id = blockDim.x * blockIdx.x + threadIdx.x;

  for (int i = 0; i < ITEMS; i++)
    shm[i][threadIdx.x] = MAKE_MTYPE(a[id]);

  for (int i = 0; i < KERNEL_INNER_REPEAT; i++) {
    MTYPE sum = MAKE_MTYPE(0);
    for (int j = 0; j < ITEMS; j++) {
      if constexpr (choose == no_conflict) {
        sum += shm[j][(threadIdx.x + 1) % 256];
      } else if constexpr (choose == boardcast) {
        sum += shm[j][threadIdx.x / 32];
      } else if constexpr (choose == multicast) {
        sum += shm[j][threadIdx.x / 4];
      } else if constexpr (choose == conflict_2_way) {
        // 0->0, 1->32, 2->1, 3->33
        sum += shm[j][((threadIdx.x % 2) * 32) + threadIdx.x / 2];
      } else if constexpr (choose == conflict_4_way) {
        // 0->0, 1->32, 2->64, 3->128;
        // 4->1, 5->33, 6->65, 7->129;
        sum += shm[j][((threadIdx.x % 4) * 32) + threadIdx.x / 4];
      } else if constexpr (choose == conflict_8_way) {
        sum += shm[j][((threadIdx.x % 8) * 32) + threadIdx.x / 8];
      }
      // else if constexpr (choose == conflict_16_way) {
      //   sum += shm[j][((threadIdx.x % 16) * 32) + threadIdx.x / 16];
      // }
    }
    shm[i % ITEMS][threadIdx.x] = sum;
  }

  a[id] = GET_MTYPE(shm[0][threadIdx.x]);
}

// Main program
int main() {
  size_t bytes = N * sizeof(float);

  float *A = (float *)malloc(bytes);

  float *d_A;
  hipMalloc(&d_A, bytes);

  for (int i = 0; i < N; i++) {
    A[i] = 1 + i / 1000.0;
  }

  hipMemcpy(d_A, A, bytes, hipMemcpyHostToDevice);

  int thr_per_blk = 256;
  int blk_in_grid = ceil(double(N) / thr_per_blk);

  printf("warm up\n");
  // warm up
  for (int i = 0; i < WARM_UP_LOOP; i++) {
    add_vectors<no_conflict><<<blk_in_grid, thr_per_blk>>>(d_A);
  }
  hipDeviceSynchronize();
  printf("finish warm up\n");

  hipMemcpy(d_A, A, bytes, hipMemcpyHostToDevice);
  float time_elapsed = 0;
  hipEvent_t start, stop;
  hipEventCreate(&start); // 创建Event
  hipEventCreate(&stop);

  hipEventRecord(start, 0);
  for (int i = 0; i < KERNEL_LOOP; i++) {
    add_vectors<no_conflict><<<blk_in_grid, thr_per_blk>>>(d_A);
  }
  hipEventRecord(stop, 0);
  hipEventSynchronize(start); // Waits for an event to complete.
  hipEventSynchronize(stop); // Waits for an event to complete.Record之前的任务
  hipEventElapsedTime(&time_elapsed, start, stop); // 计算时间差
  printf("no_conflict: time %f(ms)\n", time_elapsed);
  hipMemcpy(d_A, A, bytes, hipMemcpyHostToDevice);

  hipEventRecord(start, 0);
  for (int i = 0; i < KERNEL_LOOP; i++) {
    add_vectors<boardcast><<<blk_in_grid, thr_per_blk>>>(d_A);
  }
  hipEventRecord(stop, 0);
  hipEventSynchronize(start); // Waits for an event to complete.
  hipEventSynchronize(stop); // Waits for an event to complete.Record之前的任务
  hipEventElapsedTime(&time_elapsed, start, stop); // 计算时间差
  printf("boardcast time %f(ms)\n", time_elapsed);

  hipMemcpy(d_A, A, bytes, hipMemcpyHostToDevice);
  hipEventRecord(start, 0);
  for (int i = 0; i < KERNEL_LOOP; i++) {
    add_vectors<multicast><<<blk_in_grid, thr_per_blk>>>(d_A);
  }
  hipEventRecord(stop, 0);
  hipEventSynchronize(start); // Waits for an event to complete.
  hipEventSynchronize(stop); // Waits for an event to complete.Record之前的任务
  hipEventElapsedTime(&time_elapsed, start, stop); // 计算时间差
  printf("multicast time %f(ms)\n", time_elapsed);

  hipMemcpy(d_A, A, bytes, hipMemcpyHostToDevice);
  hipEventRecord(start, 0);
  for (int i = 0; i < KERNEL_LOOP; i++) {
    add_vectors<conflict_2_way><<<blk_in_grid, thr_per_blk>>>(d_A);
  }
  hipEventRecord(stop, 0);
  hipEventSynchronize(start); // Waits for an event to complete.
  hipEventSynchronize(stop); // Waits for an event to complete.Record之前的任务
  hipEventElapsedTime(&time_elapsed, start, stop); // 计算时间差
  printf("conflict_2_way time %f(ms)\n", time_elapsed);

  hipMemcpy(d_A, A, bytes, hipMemcpyHostToDevice);
  hipEventRecord(start, 0);
  for (int i = 0; i < KERNEL_LOOP; i++) {
    add_vectors<conflict_4_way><<<blk_in_grid, thr_per_blk>>>(d_A);
  }
  hipEventRecord(stop, 0);
  hipEventSynchronize(start); // Waits for an event to complete.
  hipEventSynchronize(stop); // Waits for an event to complete.Record之前的任务
  hipEventElapsedTime(&time_elapsed, start, stop); // 计算时间差
  printf("conflict_4_way time %f(ms)\n", time_elapsed);

  hipMemcpy(d_A, A, bytes, hipMemcpyHostToDevice);
  hipEventRecord(start, 0);
  for (int i = 0; i < KERNEL_LOOP; i++) {
    add_vectors<conflict_8_way><<<blk_in_grid, thr_per_blk>>>(d_A);
  }
  hipEventRecord(stop, 0);
  hipEventSynchronize(start); // Waits for an event to complete.
  hipEventSynchronize(stop); // Waits for an event to complete.Record之前的任务
  hipEventElapsedTime(&time_elapsed, start, stop); // 计算时间差
  printf("conflict_8_way time %f(ms)\n", time_elapsed);

  // cudaMemcpy(d_A, A, bytes, cudaMemcpyHostToDevice);
  // cudaEventRecord(start, 0);
  // for (int i = 0; i < KERNEL_LOOP; i++) {
  //   add_vectors<conflict_16_way><<<blk_in_grid, thr_per_blk>>>(d_A);
  // }
  // cudaEventRecord(stop, 0);
  // cudaEventSynchronize(start); // Waits for an event to complete.
  // cudaEventSynchronize(stop); // Waits for an event to complete.Record之前的任务
  // cudaEventElapsedTime(&time_elapsed, start, stop); // 计算时间差
  // printf("conflict_16_way time %f(ms)\n", time_elapsed);

  hipMemcpy(A, d_A, bytes, hipMemcpyDeviceToHost);

  hipMemcpy(A, d_A, bytes, hipMemcpyDeviceToHost);
  // Free CPU memory
  free(A);

  // Free GPU memory
  hipFree(d_A);

  return 0;
}
